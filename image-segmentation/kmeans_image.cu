#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/core/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/highgui/highgui_c.h>
#include <opencv2/imgcodecs/imgcodecs.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#define TPB 512
#define NLIMIT 100

/* Function to group the elements in its nearest cluster.
 * Returns the distance to said cluster. */
__device__
float grouping(const int elemID, const int dims, const int nclusters,
const float *clusters, const float *elems, int *elemClus) {
    float dist, elemDist, cPos;

    // Start by considering as part of the first cluster
    elemClus[elemID] = 0;
    elemDist = rnormf(dims, clusters);
    for (int i = 1; i < nclusters; i++) {
        // Calculate the position of the cluster in the linearized matrix
        cPos = i*dims;

        // Calculate distance
        dist = normf(dims, clusters + cPos);

        // See if it is less than the distance to the current cluster
        if (dist < elemDist) {
            elemClus[elemID] = i;
            elemDist = dist;
        }
    }

    return elemDist;
}

/* Function to update the centroid to be in the middle of the calculated
 * clusters.
 * returns: the distance moved */
__device__
float updateCentroid(const int clusID, const int dims, float *clusters,
const int nelems, const float *elems, int *elemClus) {
    float avg, dis = 0;
    int cPos = clusID*dims;

    for (int i = 0; i < dims; i++) {
        avg = 0;
        for (int j = 0; j < nelems; j++)
            if (elemClus[j] == clusID)
                avg += elems[dims*j + i]/nelems;
        dis += fabsf(clusters[cPos+1] - avg);
        clusters[cPos + i] = avg;
    }

    return dis;
}

__device__
float calcMean(const int n, const float *elems) {
    float mean = 0;
    for (int i = 0; i < n; i++) mean += elems[i]/n;
    return mean;
}

/* Function to check if the clusters moved.
 * Returns a boolean (false -> if they moved, true -> they didnt) */
__device__
int testChange(const int nclusters, const float *distances){
    for (int i = 0; i < nclusters; i++) if(distances[i] > 0) return false;
    return true;
}

/* Runs the kmean algorithm:
 * Inputs:
 *   - dims <int>: Number of dimensions
 *   - epochs <int>: Number of iterations
 *   - limit <int>: Number of updates to centroid until it gives up on finding
                the sweet spot.
 *   - nclusters <int>: Number of clusters to calculate
 *   - clusters <float*>: Pointer where all the clusters will be saved.
 *              size(float[nclusters*dims])
 *   - nelems <int>: Number of elems received
 *   - elems <float*>: Pointer to all the data. size(float[nelems*dims])
 *   - elemClus <int*>: Array where the relation elem-cluster is saved.
 *              size(int[nelems])
 *   - entropy <float*>: Pointer to where to save the calculated entropy per
 *              iteration. size(float[epochs])
 */
__global__
void kmeans(const int dims, const int epochs, const int limit,
const int nclusters, float *clusters, const int nelems, float *elems,
int *elemClus, float *entropy) {
    // shared elemDis
    __shared__ float elemDis[nelems];
    // shared distance moved
    __shared__ float movedDis[nclusters];
    // shared best clusters
    __shared__ float bestClusters[nclusters*dims];

    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    for (int i = 0; i < epochs; i++) {
        for (int j = 0; j < limit; j++) {
            // Calculate 
            if (tid < nelems)
                elemDis[tid] = grouping(tid, dims, nclusters, clusters, elems,
                    elemClus);

            __syncthreads();

            if (tid < nclusters)
                movedDis[tid] = updateCentroid(tid, dims, clusters, nelems,
                    elems, elemClus);

            __syncthreads();

            if (tid < 1) {
                // Calculate entropy
                entropy[epochs] = calcMean(nelems, elemDis);
                if (testChange(nclusters, movedDis)) break;
            }
        }
        // TODO: copy to bestCluster
    }
}

__global__
void colorClusters(const int dims, float *clusters, const int nelems,
int *elemClus, float *dst) {
    int tid = threadIdx.x + (blockIdx.x * blockDim.x), pos;

    if (tid < nelems) {
        pos = tid*dims;
        for (int i = 0; i < dims; i++)
            dst[pos+i] = clusters[dims*elemClus[tid]+i];
    }
}

int main(int argc, char *argv[]) {
    cv::Mat og, src, dst;
    int size, channels, nelems;
    int nclusters, epochs, limit;
    float *dstRaw;

    float *gpu_src, *gpu_dst, *gpu_clusters, *gpu_entropy;
    int *gpu_elemClus;

    if (argc != 5) {
        fprintf(
            stderr, 
            "usage: %s <image path> <n clusters> <epochs> <result file>\n",
            argv[0]
        );
        return -1;
    }

    // TODO: Scan number of clusters

    // TODO: Scan number of epochs

    // TODO: Scan limit

    // Scan image and convert it to float
    fprintf(stdout, "Reading image...\n");
    og = cv::imread(argv[1], cv::IMREAD_COLOR);
    og.convertTo(src, CV_32F);
    nelems = src.rows * src.cols;
    channels = src.channels();
    size = src.rows * src.cols * og.channels();

    // Allocate CPU
    fprintf(stdout, "Allocating memory in CPU...\n");
    dstRaw = (float *)malloc(sizeof(float)*size);

    // Copy to gpu
    fprintf(stdout, "Allocating memory in GPU...\n");
    hipMalloc((void**) &gpu_src, sizeof(float)*size);
    hipMalloc((void**) &gpu_clusters, sizeof(float)*nclusters*channels);
    hipMalloc((void**) &gpu_elemClus, sizeof(int)*nelems);
    hipMalloc((void**) &gpu_entropy, sizeof(float)*epochs);
    hipMalloc((void**) &gpu_dst, sizeof(float)*size);

    fprintf(stdout, "Uploading image to GPU...\n");
    hipMemcpy(gpu_src, src.data, sizeof(float)*size, hipMemcpyHostToDevice);

    // Call kmeans
    fprintf(stdout, "Applying kmeans...\n");
    kmeans<<<nelems/TPB + 1, TPB>>>(channels, epochs, limit, nclusters,
    gpu_clusters, nelems, gpu_src, gpu_elemClus, gpu_entropy);

    // Call modified image
    fprintf(stdout, "Applying colors...\n");
    colorClusters<<<nelems/TPB + 1, TPB>>>(channels, gpu_clusters, nelems,
    gpu_elemClus, gpu_dst);

    // Copy processed data to CPU
    hipMemcpy(dstRaw, gpu_dst, sizeof(float)*size, hipMemcpyDeviceToHost);

    // Convert result to opencv
    fprintf(stdout, "Obtaining image from GPU...\n");
    dst = cv::Mat(src.rows, src.cols, src.type(), dstRaw, cv::Mat::AUTO_STEP);

    // Display images
    fprintf(stdout, "Displaying images...\n");
//    cv::namedWindow("Original", cv::WINDOW_AUTOSIZE);
//    cv::imshow("Original", src);
//    cv::namedWindow("GrayScale", cv::WINDOW_AUTOSIZE);
//    cv::imshow("GrayScale", dst);
//    cv::waitKey(0);

    hipFree(gpu_src); hipFree(gpu_dst); hipFree(gpu_clusters);
    hipFree(gpu_elemClus); hipFree(gpu_entropy);
    free(dstRaw);
    return 0;
}
